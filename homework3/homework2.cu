#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

//2^29

int size = 512*1024*1024;

int* generateRandomArray(int num){
	int *result;
	result = (int*)malloc(sizeof(int) * num);
	for (int i = 0; i < num; i++){
		//result[num] = rand() % 20 - 10;
		result[i] = 1;
	}
	return result;

}

unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

double CPUtime(){
       struct timeval tp;
       gettimeofday (&tp, NULL);
       return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}



__global__ void
reduce(int *d_iarray, int *d_oarray, int n, int blockSize){
    __shared__ int sdata[256]; //hard coded for now

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    int tid = threadIdx.x;
    int i = blockIdx.x*blockSize*2 + threadIdx.x;
    int gridSize = blockSize*2*gridDim.x;

    int mySum = 0;
    
    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += d_iarray[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (i + blockSize < n)
            mySum += d_iarray[i+blockSize];

        i += gridSize;
    }
    
    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();
#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0) d_oarray[blockIdx.x] = mySum;

}
/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/

//copied from sample code, need modification

/*template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void
reduce6(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i+blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();
#endif
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}

////////////////////////////////////////////////////////////////////////////////
//! Compute sum reduction on CPU
//! We use Kahan summation for an accurate sum of large arrays.
//! http://en.wikipedia.org/wiki/Kahan_summation_algorithm
//!
//! @param data       pointer to input data
//! @param size       number of input data elements
////////////////////////////////////////////////////////////////////////////////
template<class T>
T reduceCPU(T *data, int size)
{
    T sum = data[0];
    T c = (T)0.0;

    for (int i = 1; i < size; i++)
    {
        T y = data[i] - c;
        T t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }

    return sum;
}
*/
////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the given reduction kernel
// For the kernels >= 3, we set threads / block to the minimum of maxThreads and
// n/2. For kernels < 3, we set to the minimum of maxThreads and n.  For kernel
// 6, we observe the maximum specified number of blocks, because each thread in
// that kernel can process a variable number of elements.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{

    //get device capability, to avoid block/grid size exceed the upper bound
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    if (whichKernel < 3)
    {
        threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
        blocks = (n + threads - 1) / threads;
    }
    else
    {
        threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
        //threads = 64 < 512 -> nextpow2(65/2) == 64
        blocks = (n + (threads * 2 - 1)) / (threads * 2);
        //block4 = (64 + 127)/128
    }

    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
    {
        printf("n is too large, please choose a smaller number!\n");
    }

    if (blocks > prop.maxGridSize[0])
    {
        printf("Grid size <%d> exceeds the device capability <%d>, set block size as %d (original %d)\n",
               blocks, prop.maxGridSize[0], threads*2, threads);

        blocks /= 2;
        threads *= 2;
    }

    if (whichKernel == 6)
    {
        blocks = min(maxBlocks, blocks);
    }
}


int main(){
    printf("Starting program... preparing array\n");
	int *h_array = generateRandomArray(size); //512M size
	printf("array generate complete\n");
	int *d_iarray, *d_oarray;

	int bytes = sizeof(int) * (int)size;
	int maxThreads = 256; //number of threads per block
	int maxBlocks = 64;
	int blocks = 0; //the following two should be maximum
	int threads = 0;


  	d_iarray = (int*)malloc(bytes);
	d_oarray = (int*)malloc(maxBlocks*sizeof(int));
	//alloc mem on GPU
	//int *d_array;
	printf("copy data to GPU\n");
	hipMalloc((void **)d_iarray, (size_t)bytes);
	hipMalloc((void **)d_oarray, maxBlocks * sizeof(int));

	//copy data to GPU
	hipMemcpy(d_iarray, h_array, bytes, hipMemcpyHostToDevice);
    printf("copy complete\n");
	//do the work
	
	getNumBlocksAndThreads(6, size, maxBlocks, maxThreads, blocks, threads);
	//define struct
	dim3 block(threads, 1, 1);
	dim3 grid(blocks, 1, 1);
	// when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(int) : threads * sizeof(int);
	
	printf("first round of reduction\n");
	//first round of reduction
	reduce<<< grid, block, smemSize >>>(d_iarray, d_oarray, size, 256);
	printf("complete first round\n");
	// Clear d_idata for later use as temporary buffer.
    hipMemset(d_iarray, 0, size*sizeof(int));
    
    // sum partial block sums on GPU
    int s=blocks;


    while (s > 1)
    {
        int threads = 0, blocks = 0;
        getNumBlocksAndThreads(6, s, maxBlocks, maxThreads, blocks, threads);//1 block 32 threads
        hipMemcpy(d_iarray, d_oarray, s*sizeof(int), hipMemcpyDeviceToDevice);//prepare new input date
        //reduce<T>(s, threads, blocks, kernel, d_idata, d_odata);//reduce
        
        int smemSize = (threads <= 32) ? 2 * threads * sizeof(int) : threads * sizeof(int);
        printf("second round of reduction\n");
        reduce<<< grid, block, smemSize >>>(d_iarray, d_oarray, s, 32);
        //1 block 32 threads, 
        printf("complete second round\n");


        s = (s + (threads*2-1)) / (threads*2);

        /*
        if (s > 1)
        {
            // copy result from device to host
            hipMemcpy(h_odata, d_odata, s * sizeof(T), hipMemcpyDeviceToHost);

            for (int i=0; i < s; i++)
            {
                gpu_result += h_odata[i];
            }

            needReadBack = false;
        }
        */
	}
	hipDeviceSynchronize();
	
	
	// copy final sum from device to host
	int gpu_result;
    hipMemcpy(&gpu_result, d_oarray, sizeof(int), hipMemcpyDeviceToHost);
    printf("final result is %d\n", gpu_result);
    return 0;
}


